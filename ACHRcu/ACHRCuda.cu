#include "hip/hip_runtime.h"
//STD
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <ilcplex/cplex.h>
//CUDA
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include "Utilities.cuh"
#include <hipblas.h>
#include <ctype.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
//Thrust
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <math_functions.h>
//GSL
#include <gsl/gsl_linalg.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_multifit.h>
#include <iostream>
#include <cmath>

#define EPSILON 2.2204e-16 

void computeKernelQRSeq(int nRxns, int nMets, double *S, double *h_N){
	gsl_matrix * A      = gsl_matrix_alloc(nMets,nRxns);

	//copy S in gsl format
	for(int j=0;j<nRxns;j++){
		for(int i=0;i<nMets;i++){
			gsl_matrix_set(A,i,j,S[i+j*nMets]);
		}
	}
	//declare SVD variables
	gsl_matrix * AtA  = gsl_matrix_alloc(nRxns,nRxns); 
	gsl_matrix * Q  = gsl_matrix_alloc(nRxns,nRxns); 
	gsl_matrix * R  = gsl_matrix_alloc(nRxns,nRxns); 
	gsl_vector * work = gsl_vector_alloc(nRxns);
	gsl_vector * tau = gsl_vector_alloc(nRxns);
	gsl_blas_dgemm(CblasTrans, CblasNoTrans,1.0, A, A,0.0, AtA);

	//QR of AtA
	gsl_linalg_QR_decomp(AtA, tau);
	gsl_linalg_QR_unpack (AtA, tau, Q, R);

	int k=0;
	for(int j=0;j<nRxns;j++){
		for(int i=0;i<nRxns;i++){
			h_N[k] = gsl_matrix_get(Q,i,j);
			k++;
		}
	}

}

void computeKernelQRCuda(int nRxns, int nMets, double *d_Slin, hipblasHandle_t handle, double *h_N){
	int work_size=0;
	int *devInfo;
	double *work;
	double *d_SlinTS, *d_Slin_copy;
	double alpha =1.0, beta=0.0, *d_TAU;
	gpuErrchk(hipMalloc(&d_Slin_copy, nRxns*nMets*sizeof(double)));
	gpuErrchk(hipMemcpy(d_Slin_copy,d_Slin,nRxns*nMets*sizeof(double),hipMemcpyDeviceToDevice));
	gpuErrchk(hipMalloc(&d_SlinTS, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMalloc(&d_TAU, nRxns*sizeof(double)));

        //Compute ST*S
	cublasSafeCall(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nRxns, nRxns, nMets, &alpha, d_Slin, nMets, d_Slin_copy, nMets, &beta, d_SlinTS, nRxns));

	//Init cusolver
	gpuErrchk(hipMalloc(&devInfo, sizeof(int)));
	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);
	
	//Init memory
	cusolveSafeCall(hipsolverDnDgeqrf_bufferSize(solver_handle, nRxns, nRxns, d_SlinTS,nRxns,&work_size));
	gpuErrchk(hipMalloc(&work, work_size*sizeof(double)));

	//QR
	cusolveSafeCall(hipsolverDnDgeqrf(solver_handle,nRxns,nRxns,d_SlinTS,nRxns,d_TAU,work,work_size,devInfo));
	int devInfo_h = 0;	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	if (devInfo_h != 0) std::cout	<< "Unsuccessful gerf execution\n\n";

	// --- Initializing the output Q matrix (Of course, this step could be done by a kernel function directly on the device)
	double *h_Q = (double *)malloc(nRxns*nRxns*sizeof(double));
	for(int j = 0; j < nRxns; j++)
		for(int i = 0; i < nRxns; i++)
			if (j == i) h_Q[j + i*nRxns] = 1.;
			else		h_Q[j + i*nRxns] = 0.;

	double *d_Q;			gpuErrchk(hipMalloc(&d_Q, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMemcpy(d_Q, h_Q, nRxns*nRxns*sizeof(double), hipMemcpyHostToDevice));

	cusolveSafeCall(hipsolverDnDormqr(solver_handle, HIPBLAS_SIDE_LEFT, HIPBLAS_OP_N, nRxns, nRxns, nRxns, d_SlinTS, nRxns, d_TAU, d_Q, nRxns, work, work_size, devInfo));
	
	gpuErrchk(hipMemcpy(h_N, d_Q, nRxns * nRxns * sizeof(double), hipMemcpyDeviceToHost));

	//free the memory
	hipFree(d_TAU);
	hipFree(d_Slin_copy);
	hipFree(d_Slin);
	hipsolverDnDestroy(solver_handle);
}

struct non_negative
{
    __host__ __device__
    bool operator()(const int x)
    {
        return x >= 0;
    }
};

template <typename T>
struct square
{
   __host__  __device__
        T operator()(const T& x) const { 
            return x * x;
        }
};

void computeKernelSeq(double *S,int nRxns,int nMets, double *h_N, int *istart){
	/* Sequential null space computation through full SVD with GSL, the SVD is done on AtA because
	GSL requires m>=n. N(AtA)=N(A) is the columns of V corresponding to null singular values
	Possibly a test can be done on the matrix size and the SVD can be done on At, the null 
	space would span the columns of U corresponding to null singular values*/

	//int istart;// index of the first null sv (supposdly sorted - could be tested)
	gsl_matrix * A      = gsl_matrix_alloc(nMets,nRxns);

	//copy S in gsl format
	for(int j=0;j<nRxns;j++){
		for(int i=0;i<nMets;i++){
			gsl_matrix_set(A,i,j,S[i+j*nMets]);
		}
	}
	//declare SVD variables
	gsl_matrix * AtA  = gsl_matrix_alloc(nRxns,nRxns); 
	gsl_matrix * V    = gsl_matrix_alloc(nRxns,nRxns);
	gsl_vector * sv   = gsl_vector_alloc(nRxns);
	gsl_vector * work = gsl_vector_alloc(nRxns);
	gsl_blas_dgemm(CblasTrans, CblasNoTrans,1.0, A, A,0.0, AtA);

	//SVD of AtA
	gsl_linalg_SV_decomp(AtA,V,sv,work);
	//find null sv wrt a tol (since they are sorted just find the first one)
	double tol = nRxns *  gsl_vector_max(sv) * EPSILON;
	for(int i=0;i<nRxns;i++){
		if(gsl_vector_get(sv,i) < tol){
			*istart = i;
			break;
		}
	}
	printf("sv_%d = %g\n",*istart,gsl_vector_get(sv,*istart));

	//printf("Kernel dim are %d %d\n",nRxns,nRxns-*istart);
	h_N = (double*)realloc(h_N,nRxns*(nRxns-(*istart))*sizeof(double));//Realloc h_N
	int k=0;
	for(int j=*istart;j<nRxns;j++){
		for(int i=0;i<nRxns;i++){
			h_N[k] = gsl_matrix_get(V,i,j);
			k++;
		}
	}

	//free the memory
	gsl_vector_free(sv);
	gsl_vector_free(work);
	gsl_matrix_free(AtA);
	gsl_matrix_free(V);
}

__device__ void correctBounds(double *d_ub, double *d_lb, int nRxns, double *d_prevPoint, double alpha, double beta, double *d_centerPoint, double *points, int pointsPerFile, int pointCount, int index){

	for(int i=0;i<nRxns ;i++){
		if(points[pointCount+pointsPerFile*i]>d_ub[i]){
			points[pointCount+pointsPerFile*i]=d_ub[i];
		}else if(points[pointCount+pointsPerFile*i]<d_lb[i]){
			points[pointCount+pointsPerFile*i]=d_lb[i];
		}
		d_prevPoint[nRxns*index+i]=points[pointCount+pointsPerFile*i];
		d_centerPoint[nRxns*index+i]=alpha*d_centerPoint[nRxns*index+i]+beta*points[pointCount+pointsPerFile*i];
	}
}

__global__ void reprojectPoint(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount, int index){
	int newindex = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i=newindex;i<nRxns-istart;i+=stride){
		d_umat[nRxns*index+i]=0;//d_umat now is d_tmp
		for(int j=0;j<nRxns;j++){
			d_umat[nRxns*index+i]+=d_N[j+i*nRxns]*points[pointCount+pointsPerFile*j];//here t(N)*Pt
		}
	}
}

__global__ void reprojectPoint2(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount,int index){
	int newindex= blockIdx.x * blockDim.x + threadIdx.x;
	int stride= blockDim.x * gridDim.x;

	for(int i=newindex;i<nRxns;i+=stride){
		points[pointCount+pointsPerFile*i]=0;
		for(int j=0;j<nRxns-istart;j++){
			points[pointCount+pointsPerFile*i]+=d_N[j*nRxns+i]*d_umat[nRxns*index+j];//here N*tmp
		}
	}
}

__global__ void findMaxAbs(int nRxns, double *d_umat2, int nMets, int *d_rowVec, int *d_colVec, double *d_val, int nnz, double *points, int pointsPerFile, int pointCount, int index){
	int newindex = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int k=newindex;k<nnz;k+=stride){
		d_umat2[nMets*index+d_rowVec[k]]+=d_val[k]*points[pointCount+pointsPerFile*d_colVec[k]];
	}
	
}

__device__ void advNextStep(double *d_prevPoint, double *d_umat, double d_stepDist, int nRxns, double *points, int pointsPerFile, int pointCount, int index){

	for(int i=0;i<nRxns;i++){
		points[pointCount+pointsPerFile*i]=d_prevPoint[nRxns*index+i]+d_stepDist*d_umat[nRxns*index+i];
	}
}

__device__ void fillrandPoint(double *d_fluxMat,int randpointID, int nRxns, int nPts, double *d_centerPointTmp, double *d_umat,double *d_distUb, double *d_distLb,double  *d_ub,double *d_lb,double *d_prevPoint, double d_pos, double dTol, double uTol, double d_pos_max, double d_pos_min, double *d_maxStepVec, double *d_minStepVec, double *d_min_ptr, double *d_max_ptr, int index){

	int k;
	double d_norm, init;
	k=0;
	init = 0;
	square<double>        unary_op;
    	thrust::plus<double> binary_op;

	for(int i=0;i<nRxns;i++){
		d_umat[nRxns*index+i]=d_fluxMat[i+randpointID*nRxns]-d_centerPointTmp[nRxns*index+i];//fluxMAt call is d_randPoint
	}

	d_norm=std::sqrt( thrust::transform_reduce(thrust::seq,d_umat+(nRxns*index), d_umat+(nRxns*(index+1)), unary_op, init, binary_op) );

	for(int i=0;i<nRxns;i++){
		d_umat[nRxns*index+i]=d_umat[nRxns*index+i]/d_norm;
		d_distUb[nRxns*index+i]=d_ub[i]-d_prevPoint[nRxns*index+i];
		d_distLb[nRxns*index+i]=d_prevPoint[nRxns*index+i]-d_lb[i];
		if(d_distUb[nRxns*index+i]>dTol && d_distLb[nRxns*index+i]>dTol){
			if(d_umat[nRxns*index+i] > uTol){
				d_minStepVec[2*nRxns*index+k]=-d_distLb[nRxns*index+i]/d_umat[nRxns*index+i];
				d_maxStepVec[2*nRxns*index+k]=d_distUb[nRxns*index+i]/d_umat[nRxns*index+i];
				k++;
			}else if(d_umat[nRxns*index+i] < -uTol){
				d_minStepVec[2*nRxns*index+k]=d_distUb[nRxns*index+i]/d_umat[nRxns*index+i];
				d_maxStepVec[2*nRxns*index+k]=-d_distLb[nRxns*index+i]/d_umat[nRxns*index+i];
				k++;
			}
		}
	}

	double *d_min_ptr_dev = thrust::max_element(thrust::seq,d_minStepVec+(2*nRxns*index), d_minStepVec+(2*nRxns*index) + k);
	
	double *d_max_ptr_dev = thrust::min_element(thrust::seq,d_maxStepVec+(2*nRxns*index), d_maxStepVec+(2*nRxns*index) + k);

	d_min_ptr[0] = *d_min_ptr_dev;
	d_max_ptr[0] = *d_max_ptr_dev;
}

__global__ void createRandomVec(double *randVector, int stepsPerPoint, hiprandState_t state){
        int newindex = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

	for(int i=newindex;i<stepsPerPoint;i+=stride){
		randVector[i]=(double)hiprand_uniform(&state);
	}
}

__device__ void createPoint(double *points, int stepCount, int stepsPerPoint, int nWrmup, int nRxns, hiprandState_t state, double *d_fluxMat, double *d_ub, double *d_lb, double dTol, double uTol, double maxMinTol, int pointsPerFile, int nMets, double *d_N, int istart, int totalStepCount, int pointCount, double *d_randVector, double *d_prevPoint, double *d_centerPointTmp, int *d_rowVec, int *d_colVec, double *d_val, int nnz, double *d_umat, int index, double *d_umat2, double *d_distUb, double *d_distLb, double *d_maxStepVec, double *d_minStepVec){

	int randPointId;
	double d_pos, d_pos_max, d_pos_min;
	double d_min_ptr[1], d_max_ptr[1];
	double d_stepDist, alpha, beta, dev_max[1];
	int blockSize=128, blockSize1=128, blockSize2=128;// 64 32 32
	int numBlocks=(nnz + blockSize - 1)/blockSize;
	int numBlocks1=(nRxns-istart + blockSize1 - 1)/blockSize1;
	int numBlocks2=(nRxns + blockSize2 - 1)/blockSize2;
	
	//Init min and max ptr
	d_min_ptr[0]=0;d_max_ptr[0]=0;

	while( ((abs(*d_min_ptr) < maxMinTol) && (abs(*d_max_ptr) < maxMinTol)) || (*d_min_ptr > *d_max_ptr) ){
		randPointId = ceil(nWrmup*(double)hiprand_uniform(&state));
		//printf("randPoint id is %d \n",randPointId);
		//randPointId = 9;
		fillrandPoint(d_fluxMat, randPointId, nRxns, nWrmup, d_centerPointTmp, d_umat, d_distUb, d_distLb, d_ub, d_lb, d_prevPoint, d_pos, dTol, uTol, d_pos_max, d_pos_min, d_maxStepVec, d_minStepVec, d_min_ptr, d_max_ptr, index);
		d_stepDist=(d_randVector[stepCount])*(d_max_ptr[0]-d_min_ptr[0])+d_min_ptr[0];
		//d_stepDist=(0.5)*(d_max_ptr[0]-d_min_ptr[0])+d_min_ptr[0];		//printf("min is %f max is %f step is %f \n",d_min_ptr[0],d_max_ptr[0],d_stepDist);
			//nMisses++;//Init nMisses to -1
	}

	advNextStep(d_prevPoint, d_umat, d_stepDist, nRxns, points, pointsPerFile, pointCount,index);

	if(totalStepCount % 10 == 0){
		for(int k=0;k<nMets;k++){
               		d_umat2[index*nMets+k]=0;//d_umat is d_result
       		}
		//hipDeviceSynchronize();
		findMaxAbs<<<numBlocks,blockSize>>>(nRxns, d_umat2, nMets, d_rowVec, d_colVec, d_val, nnz, points, pointsPerFile, pointCount, index);
		//hipDeviceSynchronize();
	        double *dev_max_ptr = thrust::max_element(thrust::seq,d_umat2 + (nMets*index), d_umat2 + (nMets*(index+1)));
	        dev_max[0] = *dev_max_ptr;
		if(*dev_max > 1e-9){
			hipDeviceSynchronize();
			//__syncthreads();
		        reprojectPoint<<<numBlocks2,blockSize2>>>(d_N,nRxns,istart,d_umat,points,pointsPerFile,pointCount,index);//possibly do in memory the triple mat multiplication
			hipDeviceSynchronize();
			//__syncthreads();
			reprojectPoint2<<<numBlocks1,blockSize1>>>(d_N,nRxns,istart,d_umat,points,pointsPerFile,pointCount,index);
			//__syncthreads();
			hipDeviceSynchronize();
		}
	}
	alpha=(double)(nWrmup+totalStepCount+1)/(nWrmup+totalStepCount+1+1);
	beta=1.0/(nWrmup+totalStepCount+1+1);

	//hipDeviceSynchronize();
	correctBounds(d_ub, d_lb, nRxns, d_prevPoint, alpha, beta, d_centerPointTmp,points,pointsPerFile,pointCount,index);
}

__global__ void stepPointProgress(int pointsPerFile, double *points, int stepsPerPoint, int nRxns, int nWrmup, double *d_fluxMat, double *d_ub, double *d_lb, double dTol, double uTol, double maxMinTol, int nMets, double *d_N, int istart, int *d_rowVec, int *d_colVec, double *d_val, int nnz, double *d_umat, double *d_umat2, double *d_distUb, double *d_distLb, double *d_maxStepVec, double *d_minStepVec, double *d_prevPoint, double *d_centerPointTmp, double *d_randVector){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	int stepCount, totalStepCount, pointCount;

	for(int i=index; i < pointsPerFile*stepsPerPoint; i+=stride){

		totalStepCount=index;
		stepCount =totalStepCount % stepsPerPoint;//Changed modulo by div
		pointCount=totalStepCount / stepsPerPoint;
		index= pointCount;//Always equal to point Count
		
		//printf("totalStepCount is %d stepCount %d pointCount %d index %d \n",totalStepCount,stepCount,pointCount,index);
		hiprandState_t state;
		hiprand_init(clock64(),threadIdx.x,0,&state);

		/*createRandomVec(d_randVector, stepsPerPoint, state);*///has to be fixed for every step
		createPoint(points, stepCount, stepsPerPoint, nWrmup, nRxns, state, d_fluxMat, d_ub, d_lb, dTol, uTol, maxMinTol, pointsPerFile,nMets,d_N,istart,totalStepCount,pointCount,d_randVector,d_prevPoint,d_centerPointTmp ,d_rowVec, d_colVec, d_val, nnz, d_umat,index,d_umat2,d_distUb,d_distLb,d_maxStepVec,d_minStepVec);
	}
}

int computenWrmup(char *file, int buffer){
	int iter=0;
	FILE* stream;
	char str[buffer];
	char *pt;
	stream=fopen(file,"r");
	fgets(str,buffer,stream);
	pt= strtok(str,",");
	while(pt != NULL){
		pt=strtok(NULL,",");
		iter++;
	}
	fclose(stream);
	return iter;
}

void parseLine(char *pt, int k,double *h_fluxMat,int nWrmup, int nRxns){
	int iter=0;

	while(pt != NULL){
		double a =atof(pt);
		pt = strtok(NULL,",");
		h_fluxMat[iter*nRxns+k]=a;
		iter++;		
	}
}

void createCenterPt(double *h_fluxMat, int nPts, int nRxns, double *h_centerPoint, hipblasHandle_t handle, double *d_centerPoint){
	/*Creates center point of warmup points*/
	double alpha=1.0/nPts,beta=0.0;
	double *h_v,*d_v,*d_fluxMat;

	h_v=(double*)malloc(nPts*sizeof(double));
	for(int i=0;i<nPts;i++){
		h_v[i]=1.0;
	}
	//Allocate device memory
	gpuErrchk(hipMalloc(&d_v, nPts*sizeof(double)));
	gpuErrchk(hipMalloc(&d_fluxMat, nPts*nRxns*sizeof(double)));
	gpuErrchk(hipMemcpy(d_fluxMat,h_fluxMat,nRxns*nPts*sizeof(double),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_v,h_v,nPts*sizeof(double),hipMemcpyHostToDevice));
	//do the sum
	cublasSafeCall(hipblasDgemv(handle,HIPBLAS_OP_N,nRxns,nPts,&alpha,d_fluxMat,nRxns,d_v,1,&beta,d_centerPoint,1));
	gpuErrchk(hipMemcpy(h_centerPoint,d_centerPoint,nRxns*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(d_v);
	hipFree(d_fluxMat);
}

void computeKernelCuda(double *h_Slin,int nRxns,int nMets, int *istart,double *h_N, double *d_Slin, hipblasHandle_t handle){
	/* GPU null space computation through full SVD with CUDA, the SVD is done on AtA because
	CUDA requires m>=n. N(AtA)=N(A) is the columns of V corresponding to null singular values
	Possibly a test can be done on the matrix size and the SVD can be done on At, the null 
	space would span the columns of U corresponding to null singular values*/

	// istart is the index of the first null sv (cuda sorts them)
	int work_size=0;
	int *devInfo, devInfo_h=0;
	double *d_S, *h_S, *work, *h_V, *d_U, *d_Vh, *d_V;
	double *d_SlinTS, *d_Slin_copy;
	double alpha =1.0, beta=0.0, tol;
	gpuErrchk(hipMalloc(&d_Slin_copy, nRxns*nMets*sizeof(double)));
	gpuErrchk(hipMemcpy(d_Slin_copy,d_Slin,nRxns*nMets*sizeof(double),hipMemcpyDeviceToDevice));
	gpuErrchk(hipMalloc(&d_SlinTS, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMalloc(&d_S, nRxns*sizeof(double)));
	h_S = (double*)malloc(nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_U, nRxns*nRxns*sizeof(double)));
	h_V = (double*)malloc(nRxns*nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_Vh, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMalloc(&d_V, nRxns*nRxns*sizeof(double)));

        //Compute ST*S
	cublasSafeCall(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nRxns, nRxns, nMets, &alpha, d_Slin, nMets, d_Slin_copy, nMets, &beta, d_SlinTS, nRxns));

	//SVD of ST*S because N(S)=N(ST*S) is independant of size (SVD assumes m<n)	
	gpuErrchk(hipMalloc(&devInfo, sizeof(int)));
	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);
	//CUDA SVD init
	cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, nRxns, nRxns, &work_size));
	gpuErrchk(hipMalloc(&work, work_size*sizeof(double)));
	//CUDA  SVD execution
	cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A', nRxns, nRxns, d_SlinTS, nRxns, d_S, d_U, nRxns, d_Vh, nRxns, work, work_size, NULL, devInfo));
	cublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, nRxns, nRxns, &alpha, d_Vh, nRxns, &beta, NULL, nRxns, d_V, nRxns));
	gpuErrchk(hipMemcpy(h_S,d_S,nRxns*sizeof(double),hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_V,d_V,nRxns*nRxns*sizeof(double),hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	tol = nRxns *  h_S[0] * EPSILON;
	if(devInfo_h!=0)
		printf("SVD Unsuccessful");
	for(int i=0;i<nRxns;i++){
		if(h_S[i]<tol){
			*istart=i;
			printf("h_S[%d] is %.15f \n",i,h_S[i]);
			break;
		}
	}
	int k=0;
	h_N = (double*)realloc(h_N,nRxns*(nRxns-(*istart))*sizeof(double));

	for(int j=(*istart)*nRxns;j<nRxns*nRxns;j++){
		h_N[k]=h_V[j];
		k++;
	}

	//free the memory
	hipFree(d_Slin); //keep d_Slin in memory
	hipFree(d_Slin_copy);
	hipFree(d_SlinTS);
	hipFree(d_V);
	hipFree(d_Vh);
	hipFree(d_U);
	hipFree(d_S);
	hipsolverDnDestroy(solver_handle);
	//Host
	free(h_S);
	free(h_V);
}

__global__ void fillCenterPrev(int nRxns, int pointsPerFile, double *d_centerPoint, double *d_prevPoint, double *d_centerPointTmp, double *d_randVector, int stepsPerPoint){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        int blockSize=256, numBlocks=(pointsPerFile*stepsPerPoint + blockSize - 1)/blockSize;

        for(int j=index;j<nRxns;j+=stride){
        	for(int ind=0;ind<pointsPerFile;ind++){
                	d_centerPointTmp[nRxns*ind+j]=d_centerPoint[j];
                        d_prevPoint[nRxns*ind+j]=d_centerPoint[j];//needs to be fixed wtr to prevPoint
                }       
        }       

        hiprandState_t state;
        hiprand_init(clock64(),threadIdx.x,0,&state);

        createRandomVec<<<numBlocks,blockSize>>>(d_randVector, stepsPerPoint*pointsPerFile, state);//has to be fixed for eve
}


int main(int argc, char **argv){
	double maxMinTol = 1e-9;
	double *h_fluxMat, *h_centerPoint, *d_centerPoint;
	double *cmatval;
	double *h_ub, *h_lb, *d_ub, *d_lb;
	double uTol = 1e-9, *points, *h_points, *d_umat, *d_umat2;
	double *h_Slin, *d_Slin,*h_N, *d_N, *d_fluxMat, *d_distUb, *d_distLb, *d_randVector;
	double *d_val, *h_val, *d_minStepVec, *d_maxStepVec, *d_prevPoint, *d_centerPointTmp;
	//double *d_Slin_row;
	double dTol = 1e-14;
	double elapsedTime;
	struct timespec now, tmstart;
	FILE* stream;
	char *pt;
	CPXENVptr env=NULL;
	CPXLPptr lp=NULL;
	int status, istart=0, row=0;
	int *h_rowVec, *h_colVec, *d_rowVec, *d_colVec;
	int nWrmup=0;
	int nRxns=0,nMets=0, nFiles, pointsPerFile, stepsPerPoint;
	int nzcnt,surplus,surplusbis;
	int *cmatbeg, *cmatind, totalCount;	
	int buffer = 8196*128;
	int nDevices, nnz;
	char filename[8196];
	/*All matrices are stored in column major format, except points, stored in
	row major format*/

	/*TIC*/
	clock_gettime(CLOCK_REALTIME, &tmstart);

	hipGetDeviceCount(&nDevices);
	for(int i=0; i < nDevices; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,i);
		printf("Device Number: %d\n",i);
		printf("Device name: %s\n",prop.name);
		printf("Memory clock rate (Khz): %d\n", prop.memoryClockRate);
		printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

	//Compute total step count
	nFiles=atoi(argv[3]);
	pointsPerFile=atoi(argv[4]);
	stepsPerPoint=atoi(argv[5]);
	totalCount=nFiles*pointsPerFile*stepsPerPoint;
	printf("Total count is %d \n",totalCount); 

	//Read the model
	env = CPXopenCPLEX(&status);
	printf("\nThe model supplied is %s\n", argv[1]);
	lp = CPXcreateprob(env, &status, "Problem");
	CPXreadcopyprob(env, lp, argv[1], NULL);
	CPXchgprobtype(env,lp,CPXPROB_LP);
	nMets = CPXgetnumrows (env, lp);
	nRxns = CPXgetnumcols (env, lp);
	printf("nRxns egale a %d  \n",nRxns);
	printf("nMets egale a %d  \n",nMets);	
	h_ub=(double*)calloc(nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_ub, nRxns*sizeof(double)));
	h_lb=(double*)calloc(nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_lb, nRxns*sizeof(double)));
	cmatbeg=(int*)malloc((unsigned) (nRxns+1)*sizeof(int));
	CPXgetub(env,lp,h_ub,0,nRxns-1);
	CPXgetlb(env,lp,h_lb,0,nRxns-1);
	status = CPXgetcols (env, lp, &nzcnt,cmatbeg,NULL,NULL,0,&surplus,0,nRxns-1);
	printf("the value of surplus is %d \n",surplus);
	if(status != CPXERR_NEGATIVE_SURPLUS){
		if(status!=0){
			printf("CPXgetcols for surplus failed, status =%d\n",status);
		}
		printf("All columns in range[%d, %d] are empty.\n",0,nRxns-1);
	}
	surplus=-surplus;
	cmatbeg[nRxns]=surplus;//Should find a better fix
	cmatind=(int*)malloc((unsigned) (1+surplus)*sizeof(int));
	cmatval=(double*)malloc((unsigned) (1+surplus)*sizeof(double));
	status= CPXgetcols(env,lp,&nzcnt,cmatbeg,cmatind,cmatval,surplus,&surplusbis,0,nRxns-1);
	printf("status is %d (0 means all good) \n",status);
	printf("the value of surplus is %d \n",surplusbis);
	//Initialize S
	h_Slin=(double*)calloc(nMets*nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_Slin, nRxns*nMets*sizeof(double)));
	//populate the S matrix
	nnz=0;
	for(int j=0;j<nRxns;j++){
		for(int i=cmatbeg[j];i<cmatbeg[j+1];i++){
			h_Slin[j*nMets+cmatind[i]]=cmatval[i];
			nnz++;
		}
	}
	printf("nnz is %d \n",nnz);
	//Transform into sparse format (CSR format)
	h_rowVec=(int*)calloc(nnz, sizeof(int));
	h_colVec=(int*)calloc(nnz, sizeof(int));
	h_val=(double*)calloc(nnz, sizeof(double));
	nnz=0;
	for(int i=0;i<nMets;i++){
		for(int j=0;j<nRxns;j++){
			if(h_Slin[i+j*nMets]!=0){
				h_rowVec[nnz]=i;
				h_colVec[nnz]=j;
				h_val[nnz]=h_Slin[i+j*nMets];
				nnz++;
			}
		}
	}
	printf("nnz is %d \n",nnz);
	gpuErrchk(hipMalloc(&d_rowVec, nnz*sizeof(int)));
	gpuErrchk(hipMalloc(&d_colVec, nnz*sizeof(int)));
	gpuErrchk(hipMalloc(&d_val, nnz*sizeof(double)));
	gpuErrchk(hipMemcpy(d_rowVec,h_rowVec,nnz*sizeof(int),hipMemcpyHostToDevice));	
	gpuErrchk(hipMemcpy(d_colVec,h_colVec,nnz*sizeof(int),hipMemcpyHostToDevice));	
	gpuErrchk(hipMemcpy(d_val,h_val,nnz*sizeof(double),hipMemcpyHostToDevice));	

	//Transfer ub and lb and Slin
	gpuErrchk(hipMemcpy(d_ub,h_ub,nRxns*sizeof(double),hipMemcpyHostToDevice));	
	gpuErrchk(hipMemcpy(d_lb,h_lb,nRxns*sizeof(double),hipMemcpyHostToDevice));
	CPXfreeprob(env,&lp);
	CPXcloseCPLEX(&env);

	//Read number of warmup points
	nWrmup = computenWrmup(argv[2],buffer);
	printf("nWrmup egale a %d \n",nWrmup);
        stream = fopen(argv[2],"r");
	h_fluxMat= (double*)calloc(nWrmup*nRxns, sizeof(double*));//should be init to nRxns

	//Read all points, matrix is in column-major format
	char str[buffer];
	while(fgets(str, buffer, stream)){
		pt=strtok(str,",");
		parseLine(pt,row,h_fluxMat,nWrmup,nRxns);
		row++;
	}
	//Close file
	fclose(stream);

 	//Call cublas kernel
        hipblasHandle_t handle;
        hipblasCreate(&handle);

	//Find the right null space of the S matrix
	h_N=(double*)malloc(nRxns*nRxns*sizeof(double));//Larger than actual size
	gpuErrchk(hipMemcpy(d_Slin,h_Slin,nRxns*nMets*sizeof(double),hipMemcpyHostToDevice)); //Parallel version
	computeKernelCuda(h_Slin,nRxns,nMets,&istart,h_N,d_Slin,handle);//Parallel version, based on full SVD, thus require a lot of device memory
	//computeKernelSeq(h_Slin,nRxns,nMets,h_N,&istart);//Sequential version,  much faster for models < 10k Rxns, host memory
	//istart=0;
	//computeKernelQRCuda(nRxns, nMets, d_Slin, handle, h_N);
	//computeKernelQRSeq(nRxns, nMets, h_Slin, h_N);

	//Copy the matrix
	gpuErrchk(hipMalloc(&d_N, (nRxns-istart)*nRxns*sizeof(double)));
	gpuErrchk(hipMemcpy(d_N,h_N,(nRxns-istart)*nRxns*sizeof(double), hipMemcpyHostToDevice));

	//Compute center point
	h_centerPoint = (double *)malloc(nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_centerPoint, nRxns*sizeof(double)));
	createCenterPt(h_fluxMat, nWrmup, nRxns, h_centerPoint, handle, d_centerPoint);

	//declare loop variables
	gpuErrchk(hipMalloc(&d_fluxMat, nRxns*nWrmup*sizeof(double)));
	gpuErrchk(hipMemcpy(d_fluxMat,h_fluxMat,nRxns*nWrmup*sizeof(double), hipMemcpyHostToDevice));
	//d_umat is column-major format
	int blockSize=256, numBlocks=(pointsPerFile*stepsPerPoint + blockSize - 1)/blockSize;
	int blockSize2=256, numBlocks2=(nRxns + blockSize2 - 1)/blockSize2;
	gpuErrchk(hipMalloc(&d_umat, nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_umat2, nMets*pointsPerFile*sizeof(double)));//could be removed and replaced by d_umat
	gpuErrchk(hipMalloc(&d_distUb, nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_distLb, nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_minStepVec, 2*nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_maxStepVec, 2*nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_prevPoint, nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_centerPointTmp, nRxns*pointsPerFile*sizeof(double)));
	gpuErrchk(hipMalloc(&d_randVector, pointsPerFile*stepsPerPoint*sizeof(double)));
	//could be heavily optimized use one dumat per block, use threadID, put the correct number of threads
	gpuErrchk(hipMalloc(&points, nRxns*pointsPerFile*sizeof(double)));        //Initialize prevPoint and centerPoint

	//Fill center point and previous point
	fillCenterPrev<<<numBlocks2,blockSize2>>>(nRxns, pointsPerFile,d_centerPoint,d_prevPoint,d_centerPointTmp,d_randVector,stepsPerPoint);

	//declare totalStepCount
	/*int *totalStepCount;
	gpuErrchk(hipMalloc(&totalStepCount, sizeof(int)));
	hipMemset(totalStepCount, 0 , sizeof(int));*/

        clock_gettime(CLOCK_REALTIME, &now);
        elapsedTime = (double)((now.tv_sec+now.tv_nsec*1e-9) - (double)(tmstart.tv_sec+tmstart.tv_nsec*1e-9));
        printf("Null space done in %.5f seconds.\n", elapsedTime);

	//Init total step count
	h_points=(double*)calloc(nRxns*pointsPerFile, sizeof(double));

	//Loop through files
	srand(time(NULL));
	for(int ii=0;ii<nFiles;ii++){
		printf("File %d\n",ii);
		//Initialize points matrix to zero
		hipMemset(points, 0 , nRxns*pointsPerFile*sizeof(double));
		hipDeviceSynchronize();
		stepPointProgress<<<numBlocks,blockSize>>>(pointsPerFile,points,stepsPerPoint,nRxns,nWrmup,d_fluxMat,d_ub,d_lb,dTol,uTol,maxMinTol,nMets,d_N,istart,d_rowVec, d_colVec, d_val, nnz, d_umat, d_umat2,d_distUb,d_distLb,d_maxStepVec,d_minStepVec,d_prevPoint,d_centerPointTmp,d_randVector);
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(h_points,points,nRxns*pointsPerFile*sizeof(double),hipMemcpyDeviceToHost));
		filename[0]='\0';//Init file name
		char dest[]="File";
		sprintf(filename,"%d",ii);
		strcat(dest,filename);
		//strcat(dest,".csv"); keep this one commented
		FILE * f =fopen(dest,"wb");
		int j=0;
		for(int i=0;i<nRxns;i++){
			for(j=0;j<(pointsPerFile-1);j++){
				fprintf(f,"%f,",h_points[i*pointsPerFile+j]);
			}
			fprintf(f,"%f",h_points[i*pointsPerFile+(pointsPerFile-1)]);
			fprintf(f,"\n");
		}
		fclose(f);
	}
	//printf("number of misses is %d \n",nMisses);
	//Finalize
	
        clock_gettime(CLOCK_REALTIME, &now);
        elapsedTime = (double)((now.tv_sec+now.tv_nsec*1e-9) - (double)(tmstart.tv_sec+tmstart.tv_nsec*1e-9));

        printf("Sampling done in %.5f seconds.\n", elapsedTime);
	
	//Free memory
	free(h_fluxMat);
	free(h_centerPoint);
	free(cmatval);
	free(h_ub);
	free(h_lb);
	free(h_points);
	free(h_Slin);
	free(h_N);
	free(h_colVec);
	free(h_rowVec);
	free(h_val);

	//Free cuda memory
	hipFree(d_centerPoint);
	hipblasDestroy(handle);
	hipFree(d_ub);
	hipFree(d_lb);
	hipFree(points);
	hipFree(d_umat);
	hipFree(d_umat2);
	//hipFree(d_Slin_row);
	hipFree(d_fluxMat);
	hipFree(d_N);
	//hipFree(d_Slin);
	hipFree(d_colVec);
	hipFree(d_rowVec);
	hipFree(d_minStepVec);
	hipFree(d_maxStepVec);
	hipFree(d_val);
	return 0;
}//main

