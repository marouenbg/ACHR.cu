#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <ilcplex/cplex.h>
#include <hipsolver.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include "Utilities.cuh"
#include <hipblas.h>
#include <ctype.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/extrema.h>
#include <thrust/device_ptr.h>
#include <math_functions.h>
#include <gsl/gsl_linalg.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_multifit.h>
/*#include <iostream>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cmath>*/

#define EPSILON 2.2204e-16 

void computeKernelSeq(double *S,int nRxns,int nMets, double *h_N, int *istart){
	/* Sequential null space computation through full SVD with GSL, the SVD is done on AtA because
	GSL requires m>=n. N(AtA)=N(A) is the columns of V corresponding to null singular values
	Possibly a test can be done on the matrix size and the SVD can be done on At, the null 
	space would span the columns of U corresponding to null singular values*/

	//int istart;// index of the first null sv (supposdly sorted - could be tested)
	gsl_matrix * A      = gsl_matrix_alloc(nMets,nRxns);

	//copy S in gsl format
	for(int j=0;j<nRxns;j++){
		for(int i=0;i<nMets;i++){
			gsl_matrix_set(A,i,j,S[i+j*nMets]);
		}
	}
	//declare SVD variables
	gsl_matrix * AtA  = gsl_matrix_alloc(nRxns,nRxns); 
	gsl_matrix * V    = gsl_matrix_alloc(nRxns,nRxns);
	gsl_vector * sv   = gsl_vector_alloc(nRxns);
	gsl_vector * work = gsl_vector_alloc(nRxns);
	gsl_blas_dgemm(CblasTrans, CblasNoTrans,1.0, A, A,0.0, AtA);

	//SVD of AtA
	gsl_linalg_SV_decomp(AtA,V,sv,work);
	//find null sv wrt a tol (since they are sorted just find the first one)
	double tol = nRxns *  gsl_vector_max(sv) * EPSILON;
	for(int i=0;i<nRxns;i++){
		if(gsl_vector_get(sv,i) < tol){
			*istart = i;
			break;
		}
	}
	printf("sv_%d = %g\n",*istart,gsl_vector_get(sv,*istart));

	//printf("Kernel dim are %d %d\n",nRxns,nRxns-*istart);
	h_N = (double*)realloc(h_N,nRxns*(nRxns-(*istart))*sizeof(double));//Realloc h_N
	int k=0;
	for(int j=*istart;j<nRxns;j++){
		for(int i=0;i<nRxns;i++){
			h_N[k] = gsl_matrix_get(V,i,j);
			k++;
		}
	}

	//free the memory
	gsl_vector_free(sv);
	gsl_vector_free(work);
	gsl_matrix_free(AtA);
	gsl_matrix_free(V);
}

__device__ void correctBounds(double *d_curPoint, double *d_ub, double *d_lb, int nRxns, double *d_prevPoint, double alpha, double beta, double *d_centerPoint){

	for(int i=0;i<nRxns ;i++){
		if(d_curPoint[i]>d_ub[i]){
			d_curPoint[i]=d_ub[i];
		}else if(d_curPoint[i]<d_lb[i]){
			d_curPoint[i]=d_lb[i];
		}
		d_prevPoint[i]=d_curPoint[i];
		d_centerPoint[i]=alpha*d_centerPoint[i]+beta*d_curPoint[i];
	}

}

__device__ void reprojectPoint(double *d_N, int nRxns, int istart, double *d_tmp, double *d_curPoint){

	for(int i=0;i<nRxns-istart;i++){
		d_tmp[i]=0;
		for(int j=0;j<nRxns;j++){
			d_tmp[i]+=d_N[j+i*nRxns]*d_curPoint[j];//here t(N)*Pt
		}
	}
	
	for(int i=0;i<nRxns;i++){
		d_curPoint[i]=0;
		for(int j=0;j<nRxns-istart;j++){
			d_curPoint[i]+=d_N[j*nRxns+i]*d_tmp[j];//here N*tmp
		}
	}
}

__device__ void findMaxAbs(int nRxns, double *d_curPoint, double *d_result, int nMets, double *d_Slin_row, double *dev_max){

	for(int i=0;i<nMets;i++){
		d_result[i]=0;
		for(int j=0;j<nRxns;j++){
			d_result[i]+=d_Slin_row[j+i*nRxns]*d_curPoint[j];
		}
		d_result[i]=abs(d_result[i]);
	}

	double *dev_max_ptr = thrust::max_element(thrust::device,d_result, d_result + nMets);
	dev_max[0] = *dev_max_ptr;
	
}

__device__ void addPoint(int pointCount,double *points,double *d_curPoint, int pointsPerFile, int nRxns){
	for(int i=0;i<nRxns;i++){//in row major format (everything else is column major)
			points[pointCount+pointsPerFile*i]=d_curPoint[i];
		}
}

__device__ void advNextStep(double *d_prevPoint, double *d_curPoint, double *d_u, double d_stepDist, int nRxns){
	for(int i=0;i<nRxns;i++){
		d_curPoint[i]=d_prevPoint[i]+d_stepDist*d_u[i];
	}
}

__device__ void fillrandPoint(double *d_fluxMat,int randpointID, int nRxns, int nPts, double *d_centerPoint, double *d_u,double *d_distUb, double *d_distLb,double  *d_ub,double *d_lb,double *d_prevPoint, int d_nValid,double d_pos, double dTol, double uTol, double d_pos_max, double d_pos_min, double *d_maxStepVec, double *d_minStepVec, double *d_min_ptr, double *d_max_ptr){

	int k;
	double d_norm, d_sum;
	k=0;
	/*square<double>        unary_op;
    	thrust::plus<double> binary_op;
    	double init = 0;*/

	for(int i=0;i<nRxns;i++){
		d_u[i]=d_fluxMat[i+randpointID*nRxns]-d_centerPoint[i];//fluxMAt call is d_randPoint
		d_sum+=pow(d_u[i],2);//maybe save square in unsued vector and sum faster with thrust
	}

	//d_sum = thrust::reduce(thrust::device, d_u2, d_u2+nRxns);//MAYBE JUST increment d_sum instead of thrust
	//d_norm=std::sqrt( thrust::transform_reduce(d_u, d_u+nRxns, unary_op, init, binary_op) );
	d_norm=sqrt(d_sum);
	
	for(int i=0;i<nRxns;i++){
		d_u[i]=d_u[i]/d_norm;
		d_distUb[i]=d_ub[i]-d_prevPoint[i];
		d_distLb[i]=d_prevPoint[i]-d_lb[i];
		if(d_distUb[i]>dTol && d_distLb[i]>dTol){
			if(d_u[i] > uTol){
				d_minStepVec[k]=-d_distLb[i]/d_u[i];
				d_maxStepVec[k]=d_distUb[i]/d_u[i];
				k++;
			}else if(d_u[i] < -uTol){
				d_minStepVec[k]=d_distUb[i]/d_u[i];
				d_maxStepVec[k]=-d_distLb[i]/d_u[i];
				k++;
			}
			d_nValid++;
		}
	}

	double *d_min_ptr_dev = thrust::max_element(thrust::device,d_minStepVec, d_minStepVec + k);
	
	double *d_max_ptr_dev = thrust::min_element(thrust::device,d_maxStepVec, d_maxStepVec + k);

	d_min_ptr[0] = *d_min_ptr_dev;
	d_max_ptr[0] = *d_max_ptr_dev;
}

__device__ void createRandomVec(double *randVector, int stepsPerPoint, hiprandState_t state){
	for(int i=0;i<stepsPerPoint;i++){
		randVector[i]=(double)hiprand_uniform(&state);
	}
}

__device__ void createPoint(double *points, int stepCount, int stepsPerPoint, int nWrmup, int nRxns,hiprandState_t state, double *d_fluxMat, double *d_ub, double *d_lb, double dTol, double uTol, double maxMinTol, int pointsPerFile, int nMets, double *d_Slin_row, double *d_N, int istart, double *d_centerPoint, int totalStepCount, int pointCount, double *d_randVector, double *d_prevPoint, double *d_centerPointTmp){
	
	int randPointId, d_nValid;
	double d_u[1100];
	double d_distUb[1100];
	double d_distLb[1100];
	double d_curPoint[1100];
	//double d_result[1100];becomes d_distUb
	//double d_tmp[1100];becomes d_distLB
	double d_maxStepVec[2200];
	double d_minStepVec[2200];
	double d_pos, d_pos_max, d_pos_min;
	double d_min_ptr[1], d_max_ptr[1];
	double d_stepDist, dev_max[1], alpha, beta;

	d_nValid=0;
	while(stepCount < stepsPerPoint){
		randPointId = ceil(nWrmup*(double)hiprand_uniform(&state));
		//printf("randPoint id is %d \n",randPointId);
		//randPointId = 9;
		fillrandPoint(d_fluxMat, randPointId, nRxns, nWrmup, d_centerPointTmp, d_u, d_distUb, d_distLb, d_ub, d_lb, d_prevPoint, d_nValid,d_pos, dTol, uTol, d_pos_max, d_pos_min, d_maxStepVec, d_minStepVec, d_min_ptr, d_max_ptr);
		d_stepDist=(d_randVector[stepCount])*(d_max_ptr[0]-d_min_ptr[0])+d_min_ptr[0];
		//d_stepDist=(0.5)*(d_max_ptr[0]-d_min_ptr[0])+d_min_ptr[0];
		//printf("min is %f max is %f step is %f \n",d_min_ptr[0],d_max_ptr[0],d_stepDist);
		if ( ((abs(*d_min_ptr) < maxMinTol) && (abs(*d_max_ptr) < maxMinTol)) || (*d_min_ptr > *d_max_ptr) ){ 
			//nMisses++;
			continue;
		}
		advNextStep(d_prevPoint, d_curPoint, d_u, d_stepDist,nRxns);
		if(totalStepCount % 10 == 0){
			findMaxAbs(nRxns, d_curPoint, d_distUb, nMets, d_Slin_row, dev_max);
			if(*dev_max > 1e-9){
				reprojectPoint(d_N,nRxns,istart,d_distLb,d_curPoint);//possibly do in memory the triple mat multiplication
			}
		}
		
		alpha=(double)(nWrmup+totalStepCount+1)/(nWrmup+totalStepCount+1+1);
		beta=1.0/(nWrmup+totalStepCount+1+1);
		
		correctBounds(d_curPoint, d_ub, d_lb, nRxns, d_prevPoint, alpha, beta, d_centerPointTmp);
		
		stepCount++;
		totalStepCount++;
	}
	addPoint(pointCount, points, d_curPoint, pointsPerFile, nRxns);
	
}

__global__ void stepPointProgress(double *d_Slin_row,int pointsPerFile, double *points, int stepsPerPoint, int nRxns, int nWrmup, double *d_fluxMat, double *d_ub, double *d_lb, double dTol, double uTol, double maxMinTol, int nMets, double *d_N, int istart, double *d_centerPoint){
	int index = blockIdx.x * blockDim.x +threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	if(index < pointsPerFile){
		int stepCount, totalStepCount;
		double d_prevPoint[1100], d_centerPointTmp[1100], d_randVector[1100];

		hiprandState_t state;
		hiprand_init(clock64(),threadIdx.x,0,&state);

		stepCount=0;
		totalStepCount=0;

		for(int i=0;i<nRxns;i++){
			d_centerPointTmp[i]=d_centerPoint[i];
			d_prevPoint[i]=d_centerPoint[i];
		}

		for(int pointCount=index;pointCount<pointsPerFile;pointCount+=stride){
			createRandomVec(d_randVector, stepsPerPoint, state);
			createPoint(points, stepCount, stepsPerPoint, nWrmup, nRxns, state, d_fluxMat, d_ub, d_lb, dTol, uTol, maxMinTol, pointsPerFile,nMets,d_Slin_row,d_N,istart,d_centerPoint,totalStepCount,pointCount,d_randVector,d_prevPoint,d_centerPointTmp);
		}
	}
}

int computenWrmup(char *file, int buffer){
	int iter=0;
	FILE* stream;
	char str[buffer];
	char *pt;
	stream=fopen(file,"r");
	fgets(str,buffer,stream);
	pt= strtok(str,",");
	while(pt != NULL){
		pt=strtok(NULL,",");
		iter++;
	}
	fclose(stream);
	return iter;
}

void parseLine(char *pt, int k,double *h_fluxMat,int nWrmup, int nRxns){
	int iter=0;

	while(pt != NULL){
		double a =atof(pt);
		pt = strtok(NULL,",");
		h_fluxMat[iter*nRxns+k]=a;
		iter++;		
	}
}

void createCenterPt(double *h_fluxMat, int nPts, int nRxns, double *h_centerPoint, hipblasHandle_t handle, double *d_centerPoint){
	/*Creates center point of warmup points*/
	double alpha=1.0/nPts,beta=0.0;
	double *h_v,*d_v,*d_fluxMat;

	h_v=(double*)malloc(nPts*sizeof(double));
	for(int i=0;i<nPts;i++){
		h_v[i]=1.0;
	}
	//Allocate device memory
	gpuErrchk(hipMalloc(&d_v, nPts*sizeof(double)));
	gpuErrchk(hipMalloc(&d_fluxMat, nPts*nRxns*sizeof(double)));
	gpuErrchk(hipMemcpy(d_fluxMat,h_fluxMat,nRxns*nPts*sizeof(double),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_v,h_v,nPts*sizeof(double),hipMemcpyHostToDevice));
	//do the sum
	cublasSafeCall(hipblasDgemv(handle,HIPBLAS_OP_N,nRxns,nPts,&alpha,d_fluxMat,nRxns,d_v,1,&beta,d_centerPoint,1));
	gpuErrchk(hipMemcpy(h_centerPoint,d_centerPoint,nRxns*sizeof(double),hipMemcpyDeviceToHost));

	hipFree(d_v);
	hipFree(d_fluxMat);
}

void computeKernelCuda(double *h_Slin,int nRxns,int nMets, int *istart,double *h_N, double *d_Slin, hipblasHandle_t handle){
	/* GPU null space computation through full SVD with CUDA, the SVD is done on AtA because
	CUDA requires m>=n. N(AtA)=N(A) is the columns of V corresponding to null singular values
	Possibly a test can be done on the matrix size and the SVD can be done on At, the null 
	space would span the columns of U corresponding to null singular values*/

	// istart is the index of the first null sv (cuda sorts them)
	int work_size=0;
	int *devInfo, devInfo_h=0;
	double *d_S, *h_S, *work, *h_V, *d_U, *d_Vh, *d_V;
	double *d_SlinTS, *d_Slin_copy;
	double alpha =1.0, beta=0.0, tol;
	gpuErrchk(hipMalloc(&d_Slin_copy, nRxns*nMets*sizeof(double)));
	gpuErrchk(hipMemcpy(d_Slin_copy,d_Slin,nRxns*nMets*sizeof(double),hipMemcpyDeviceToDevice));
	gpuErrchk(hipMalloc(&d_SlinTS, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMalloc(&d_S, nRxns*sizeof(double)));
	h_S = (double*)malloc(nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_U, nRxns*nRxns*sizeof(double)));
	h_V = (double*)malloc(nRxns*nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_Vh, nRxns*nRxns*sizeof(double)));
	gpuErrchk(hipMalloc(&d_V, nRxns*nRxns*sizeof(double)));

        //Compute ST*S
	cublasSafeCall(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nRxns, nRxns, nMets, &alpha, d_Slin, nMets, d_Slin_copy, nMets, &beta, d_SlinTS, nRxns));
	hipFree(d_Slin); //keep d_Slin in memory
	hipFree(d_Slin_copy);

	//SVD of ST*S because N(S)=N(ST*S) is independant of size (SVD assumes m<n)	
	gpuErrchk(hipMalloc(&devInfo, sizeof(int)));
	hipsolverHandle_t solver_handle;
	hipsolverDnCreate(&solver_handle);
	//CUDA SVD init
	cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, nRxns, nRxns, &work_size));
	gpuErrchk(hipMalloc(&work, work_size*sizeof(double)));
	//CUDA  SVD execution
	cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A', nRxns, nRxns, d_SlinTS, nRxns, d_S, d_U, nRxns, d_Vh, nRxns, work, work_size, NULL, devInfo));
	cublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_N, nRxns, nRxns, &alpha, d_Vh, nRxns, &beta, NULL, nRxns, d_V, nRxns));
	gpuErrchk(hipMemcpy(h_S,d_S,nRxns*sizeof(double),hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_V,d_V,nRxns*nRxns*sizeof(double),hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
	tol = nRxns *  h_S[0] * EPSILON;
	if(devInfo_h!=0)
		printf("SVD Unsuccessful");
	for(int i=0;i<nRxns;i++){
		if(h_S[i]<tol){
			*istart=i;
			printf("h_S[%d] is %.15f \n",i,h_S[i]);
			break;
		}
	}
	int k=0;
	h_N = (double*)realloc(h_N,nRxns*(nRxns-(*istart))*sizeof(double));

	for(int j=(*istart)*nRxns;j<nRxns*nRxns;j++){
		h_N[k]=h_V[j];
		k++;
	}

	//free the memory
	hipFree(d_SlinTS);
	hipFree(d_V);
	hipFree(d_Vh);
	hipFree(d_U);
	hipFree(d_S);
	hipsolverDnDestroy(solver_handle);
}

int main(int argc, char **argv){
	double maxMinTol = 1e-9;
	double *h_fluxMat, *h_centerPoint, *d_centerPoint;
	double *cmatval;
	double *h_ub, *h_lb, *d_ub, *d_lb;
	double uTol = 1e-9, *points, *h_points;
	double *h_Slin, *d_Slin,*h_N, *d_N, *d_fluxMat, *d_Slin_row;
	double dTol = 1e-14;
	double elapsedTime;
	struct timespec now, tmstart;
	FILE* stream;
	char *pt;
	CPXENVptr env=NULL;
	CPXLPptr lp=NULL;
	int status, istart=0, row=0;
	int nWrmup=0;
	int nRxns=0,nMets=0, nFiles, pointsPerFile, stepsPerPoint;
	int nzcnt,surplus,surplusbis;
	int *cmatbeg, *cmatind, totalCount;	
	int buffer = 8196*128;
	int nDevices;
	char filename[8196];
	/*All matrices are stored in column major format, except points, stored in
	row major format*/

	/*TIC*/
	clock_gettime(CLOCK_REALTIME, &tmstart);

	hipGetDeviceCount(&nDevices);
	for(int i=0; i < nDevices; i++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,i);
		printf("Device Number: %d\n",i);
		printf("Device name: %s\n",prop.name);
		printf("Memory clock rate (Khz): %d\n", prop.memoryClockRate);
		printf("Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
	}

	//Compute total step count
	nFiles=atoi(argv[3]);
	pointsPerFile=atoi(argv[4]);
	stepsPerPoint=atoi(argv[5]);
	totalCount=nFiles*pointsPerFile*stepsPerPoint;
	printf("Total count is %d \n",totalCount); 

	//Read the model
	env = CPXopenCPLEX(&status);
	printf("\nThe model supplied is %s\n", argv[1]);
	lp = CPXcreateprob(env, &status, "Problem");
	CPXreadcopyprob(env, lp, argv[1], NULL);
	CPXchgprobtype(env,lp,CPXPROB_LP);
	nMets = CPXgetnumrows (env, lp);
	nRxns = CPXgetnumcols (env, lp);
	printf("nRxns egale a %d  \n",nRxns);
	printf("nMets egale a %d  \n",nMets);	
	h_ub=(double*)calloc(nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_ub, nRxns*sizeof(double)));
	h_lb=(double*)calloc(nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_lb, nRxns*sizeof(double)));
	cmatbeg=(int*)malloc((unsigned) (nRxns+1)*sizeof(int));
	CPXgetub(env,lp,h_ub,0,nRxns-1);
	CPXgetlb(env,lp,h_lb,0,nRxns-1);
	status = CPXgetcols (env, lp, &nzcnt,cmatbeg,NULL,NULL,0,&surplus,0,nRxns-1);
	printf("the value of surplus is %d \n",surplus);
	if(status != CPXERR_NEGATIVE_SURPLUS){
		if(status!=0){
			printf("CPXgetcols for surplus failed, status =%d\n",status);
		}
		printf("All columns in range[%d, %d] are empty.\n",0,nRxns-1);
	}
	surplus=-surplus;
	cmatbeg[nRxns]=surplus;//Should find a better fix
	cmatind=(int*)malloc((unsigned) (1+surplus)*sizeof(int));
	cmatval=(double*)malloc((unsigned) (1+surplus)*sizeof(double));
	status= CPXgetcols(env,lp,&nzcnt,cmatbeg,cmatind,cmatval,surplus,&surplusbis,0,nRxns-1);
	printf("status is %d (0 means all good) \n",status);
	printf("the value of surplus is %d \n",surplusbis);
	//Initialize S
	h_Slin=(double*)calloc(nMets*nRxns, sizeof(double));
	gpuErrchk(hipMalloc(&d_Slin, nRxns*nMets*sizeof(double)));
	//populate the S matrix
	for(int j=0;j<nRxns;j++){
		for(int i=cmatbeg[j];i<cmatbeg[j+1];i++){
			h_Slin[j*nMets+cmatind[i]]=cmatval[i];
		}
	}
	
	//Transfer ub and lb and Slin
	gpuErrchk(hipMemcpy(d_ub,h_ub,nRxns*sizeof(double),hipMemcpyHostToDevice));	
	gpuErrchk(hipMemcpy(d_lb,h_lb,nRxns*sizeof(double),hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_Slin,h_Slin,nRxns*nMets*sizeof(double),hipMemcpyHostToDevice));
	CPXfreeprob(env,&lp);
	CPXcloseCPLEX(&env);

	//Read number of warmup points
	nWrmup = computenWrmup(argv[2],buffer);
	printf("nWrmup egale a %d \n",nWrmup);
        stream = fopen(argv[2],"r");
	h_fluxMat= (double*)calloc(nWrmup*nRxns, sizeof(double*));//should be init to nRxns

	//Read all points, matrix is in column-major format
	char str[buffer];
	while(fgets(str, buffer, stream)){
		pt=strtok(str,",");
		parseLine(pt,row,h_fluxMat,nWrmup,nRxns);
		row++;
	}
	//Close file
	fclose(stream);

 	//Call cublas kernel
        hipblasHandle_t handle;
        hipblasCreate(&handle);

	//Find the right null space of the S matrix
	h_N=(double*)malloc(nRxns*nRxns*sizeof(double));//Larger than actual size
	//computeKernelCuda(h_Slin,nRxns,nMets,&istart,h_N,d_Slin,handle);//Parallel version, based on full SVD, thus require a lot of device memory
	computeKernelSeq(h_Slin,nRxns,nMets,h_N,&istart);//Sequential version,  much faster for models < 10k Rxns, host memory
	gpuErrchk(hipMalloc(&d_N, (nRxns-istart)*nRxns*sizeof(double)));
	gpuErrchk(hipMemcpy(d_N,h_N,(nRxns-istart)*nRxns*sizeof(double), hipMemcpyHostToDevice));
	double alpha=1.0, beta=0.0;

	//declare d_slin_row
	gpuErrchk(hipMalloc(&d_Slin_row, nRxns*nMets*sizeof(double)));
	cublasSafeCall(hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, nRxns, nMets, &alpha, d_Slin, nMets, &beta, NULL, nRxns, d_Slin_row, nRxns));
	gpuErrchk(hipMemcpy(h_Slin,d_Slin_row,nMets*nRxns*sizeof(double),hipMemcpyDeviceToHost));

	//Compute center point
	h_centerPoint = (double *)malloc(nRxns*sizeof(double));
	gpuErrchk(hipMalloc(&d_centerPoint, nRxns*sizeof(double)));
	createCenterPt(h_fluxMat, nWrmup, nRxns, h_centerPoint, handle, d_centerPoint);

	//declare loop variables
	gpuErrchk(hipMalloc(&d_fluxMat, nRxns*nWrmup*sizeof(double)));
	gpuErrchk(hipMemcpy(d_fluxMat,h_fluxMat,nRxns*nWrmup*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMalloc(&points, nRxns*pointsPerFile*sizeof(double)));

        clock_gettime(CLOCK_REALTIME, &now);
        elapsedTime = (double)((now.tv_sec+now.tv_nsec*1e-9) - (double)(tmstart.tv_sec+tmstart.tv_nsec*1e-9));
        printf("Null space done in %.5f seconds.\n", elapsedTime);

	//Init total step count
	h_points=(double*)calloc(nRxns*pointsPerFile, sizeof(double));
	int blockSize=64, numBlocks=(pointsPerFile + blockSize - 1)/blockSize;

	//Loop through files
	srand(time(NULL));
	for(int ii=0;ii<nFiles;ii++){
		printf("File %d\n",ii);
		//Initialize points matrix to zero
		hipMemset(points, 0 , nRxns*pointsPerFile*sizeof(double));
		stepPointProgress<<<numBlocks, blockSize>>>(d_Slin_row,pointsPerFile,points,stepsPerPoint,nRxns,nWrmup,d_fluxMat,d_ub,d_lb,dTol,uTol,maxMinTol,nMets,d_N,istart,d_centerPoint);
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(h_points,points,nRxns*pointsPerFile*sizeof(double),hipMemcpyDeviceToHost));
		filename[0]='\0';//Init file name
		char dest[]="File";
		sprintf(filename,"%d",ii);
		strcat(dest,filename);
		//strcat(dest,".csv"); keep this one commented
		FILE * f =fopen(dest,"wb");
		int j=0;
		for(int i=0;i<nRxns;i++){
			for(j=0;j<(pointsPerFile-1);j++){
				fprintf(f,"%f,",h_points[i*pointsPerFile+j]);
			}
			fprintf(f,"%f",h_points[i*pointsPerFile+(pointsPerFile-1)]);
			fprintf(f,"\n");
		}
		fclose(f);
	}
	//printf("number of misses is %d \n",nMisses);
	//Finalize
	
        clock_gettime(CLOCK_REALTIME, &now);
        elapsedTime = (double)((now.tv_sec+now.tv_nsec*1e-9) - (double)(tmstart.tv_sec+tmstart.tv_nsec*1e-9));

        printf("Sampling done in %.5f seconds.\n", elapsedTime);
	
	//Free memory
	free(h_fluxMat);
	free(h_centerPoint);
	free(cmatval);
	free(h_ub);
	free(h_lb);
	free(h_points);
	free(h_Slin);
	free(h_N);

	//Free cuda memory
	hipFree(d_centerPoint);
	hipblasDestroy(handle);
	hipFree(d_ub);
	hipFree(d_lb);
	hipFree(points);
	hipFree(d_Slin_row);
	hipFree(d_fluxMat);
	hipFree(d_N);
	//hipFree(d_Slin);

	return 0;
}//main

